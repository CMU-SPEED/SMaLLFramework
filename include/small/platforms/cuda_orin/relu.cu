#include "hip/hip_runtime.h"
//****************************************************************************
// SMaLL, Software for Machine Learning Libraries
// Copyright 2023 by The SMaLL Contributors, All Rights Reserved.
// SPDX-License-Identifier: BSD-3-Clause
//
// For additional details (including references to third party source code and
// other files) see the LICENSE file or contact permission@sei.cmu.edu. See
// Contributors.txt for a full list of contributors. Created, in part, with
// funding and support from the U.S. Government (see Acknowledgments.txt file).
// DM23-0126
//****************************************************************************

#include <hip/hip_runtime.h>

//****************************************************************************

__global__ void ReLU_ker(int N, value_type const* in, value_type* out)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x)
    {
      out[i] = (in[i] > 0.0 ? in[i] : 0);
    }
}
